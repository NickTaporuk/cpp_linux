#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
#include<stdlib.h>

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)

#define printf(f, ...) ((void)(f, __VA_ARGS__),0)

#endif

__global__ void helloFromGPU (void)
{
    printf("Hello World from GPU!\n");
}

int main(void)
{
// hello from cpu
printf("Hello World from CPU!\n");
helloFromGPU <<<1, 10>>>();
hipDeviceReset();
return 0;
}
